
#include <hip/hip_runtime.h>
#include <stdio.h>



__global__ void printHelloWorld() {

    printf("Hello World from GPU!\n");

}



int main() {

    printHelloWorld<<<1, 1>>>(); // Launch kernel with 1 block and 1 thread

    hipDeviceSynchronize(); // Wait for GPU to finish

    return 0;

}
